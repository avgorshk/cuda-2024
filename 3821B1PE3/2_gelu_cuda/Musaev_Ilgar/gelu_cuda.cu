#include "hip/hip_runtime.h"
// Copyright (c) 2024 Musaev Ilgar
#include "gelu_cuda.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void geluKernel(const float* input, float* output, size_t s) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < s) {
        float x = input[i];
        float x_three = x * x * x;
        float tanh = sqrtf(2.0f / M_PI) * (x + 0.044715f * x_three);
        output[i] = 0.5f * x * (1.0f + tanhf(tanh));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t n = input.size();
    std::vector<float> output(n);
    float* d_input = nullptr;
    float* d_output = nullptr;
    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));
    hipMemcpy(d_input, input.data(), n * sizeof(float), hipMemcpyHostToDevice);
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    geluKernel<<<numBlocks, blockSize>>>(d_input, d_output, n);
    hipMemcpy(output.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    return output;
}