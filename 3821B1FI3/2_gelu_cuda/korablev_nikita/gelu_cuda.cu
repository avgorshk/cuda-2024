#include "hip/hip_runtime.h"
// Copyright (c) 2024 Korablev Nikita
#include "gelu_cuda.h"
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <>

__global__ void GeluKernel(const float* input, float* output, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanhf(sqrtf(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    size_t size = input.size();
    std::vector<float> output(size);

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

    auto blockSize = deviceProp.maxThreadsPerBlock;
    auto blockCount = (size + blockSize - 1) / blockSize;

    GeluKernel<<<blockCount, blockSize>>>(d_input, d_output, size);

    hipMemcpy(output.data(), d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
